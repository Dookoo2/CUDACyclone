#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <cstdint>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <iostream>
#include <iomanip>
#include <sstream>
#include <string>
#include <thread>
#include <chrono>
#include <cmath>
#include "CUDAMath.h"
#include "sha256.h"
#include "CUDAHash.cuh"
#include "CUDAUtils.h"
#include "CUDAStructures.h"

__device__ __forceinline__ int load_found_flag_relaxed(const int* p) {
    return *((const volatile int*)p);
}

__device__ __forceinline__ bool warp_found_ready(const int* __restrict__ d_found_flag,
                                                 unsigned full_mask,
                                                 unsigned lane)
{
    int f = 0;
    if (lane == 0) {
        f = load_found_flag_relaxed(d_found_flag);
    }
    f = __shfl_sync(full_mask, f, 0);
    return f == FOUND_READY;
}

#ifndef MAX_BATCH_SIZE
#define MAX_BATCH_SIZE 1024
#endif
#ifndef WARP_SIZE
#define WARP_SIZE 32
#endif

__device__ uint64_t g_pGx[MAX_BATCH_SIZE * 4];
__device__ uint64_t g_pGy[MAX_BATCH_SIZE * 4];

__device__ uint64_t g_Jx[4];
__device__ uint64_t g_Jy[4];

__device__ __forceinline__ uint64_t to_u64_clamped(const uint64_t a[4]) {
     return a[0];
}

// Last try or i will burn this shit
__launch_bounds__(256, 2)
__global__ void kernel_point_add_and_check_sliced(
    const uint64_t* __restrict__ Px,          
    const uint64_t* __restrict__ Py,          
    uint64_t* __restrict__ Rx,                
    uint64_t* __restrict__ Ry,                
    uint64_t* __restrict__ start_scalars,     
    uint64_t* __restrict__ counts256,         
    uint64_t threadsTotal,
    uint32_t batch_size,                     
    uint32_t max_batches_per_launch,          
    int do_initial_anchor_check,              
    int* __restrict__ d_found_flag,
    FoundResult* __restrict__ d_found_result,
    unsigned long long* __restrict__ hashes_accum,
    unsigned int* __restrict__ d_any_left
)
{
    const int batch = (int)batch_size;
    if (batch <= 0 || (batch & 1)) return;
    if (batch > MAX_BATCH_SIZE) return;
    const int half = batch >> 1;

    const uint64_t gid = (uint64_t)blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= threadsTotal) return;

    const unsigned lane      = (unsigned)(threadIdx.x & (WARP_SIZE - 1));
    const unsigned full_mask = 0xFFFFFFFFu;

    if (warp_found_ready(d_found_flag, full_mask, lane)) return;

    const uint32_t target_prefix = c_target_prefix;

    unsigned int local_hashes = 0;
    #define FLUSH_THRESHOLD 16384u
    #define WARP_FLUSH_HASHES()                                                              \
        do {                                                                                 \
            unsigned long long v = warp_reduce_add_ull((unsigned long long)local_hashes);    \
            if (lane == 0 && v) atomicAdd(hashes_accum, v);                                  \
            local_hashes = 0;                                                                \
        } while (0)
    #define MAYBE_WARP_FLUSH()                                                               \
        do { if ((local_hashes & (FLUSH_THRESHOLD - 1u)) == 0u) WARP_FLUSH_HASHES(); } while (0)

    uint64_t x1[4], y1[4], base_scalar[4];
#pragma unroll
    for (int i = 0; i < 4; ++i) {
        const uint64_t idx = gid * 4 + i;
        x1[i] = Px[idx];
        y1[i] = Py[idx];
        base_scalar[i] = start_scalars[idx];
    }

    uint64_t rem[4];
#pragma unroll
    for (int i = 0; i < 4; ++i) rem[i] = counts256[gid*4 + i];

     if ((rem[0] | rem[1] | rem[2] | rem[3]) == 0ull) {
#pragma unroll
        for (int i = 0; i < 4; ++i) {
            Rx[gid*4+i] = x1[i];
            Ry[gid*4+i] = y1[i];
        }
        WARP_FLUSH_HASHES();
        return;
    }

    // First anchor try
    if (do_initial_anchor_check) {
        uint8_t tmp_hash[20];
        uint8_t prefix = (uint8_t)(y1[0] & 1ULL) ? 0x03 : 0x02;
        getHash160_33_from_limbs(prefix, x1, tmp_hash);
        ++local_hashes; MAYBE_WARP_FLUSH();

        bool local_pref = hash160_prefix_equals(tmp_hash, target_prefix);
        if (__any_sync(full_mask, local_pref)) {
            if (local_pref && hash160_matches_prefix_then_full(tmp_hash, c_target_hash160, target_prefix)) {
                if (atomicCAS(d_found_flag, FOUND_NONE, FOUND_LOCK) == FOUND_NONE) {
                    d_found_result->threadId = (int)gid;
                    d_found_result->iter     = 0;
#pragma unroll
                    for (int k = 0; k < 4; ++k) d_found_result->scalar[k] = base_scalar[k];
#pragma unroll
                    for (int k = 0; k < 4; ++k) d_found_result->Rx[k] = x1[k];
#pragma unroll
                    for (int k = 0; k < 4; ++k) d_found_result->Ry[k] = y1[k];
                    __threadfence_system();
                    atomicExch(d_found_flag, FOUND_READY);
                }
            }
            __syncwarp(full_mask);
            WARP_FLUSH_HASHES();
            return;
        }

        sub256_u64_inplace(rem, 1ull);
        if ((rem[0] | rem[1] | rem[2] | rem[3]) == 0ull) {
#pragma unroll
            for (int i = 0; i < 4; ++i) {
                Rx[gid*4+i] = x1[i];
                Ry[gid*4+i] = y1[i];
                counts256[gid*4+i] = rem[i];
                start_scalars[gid*4+i] = base_scalar[i];
            }
            WARP_FLUSH_HASHES();
            return;
        }
    }

    uint32_t batches_done = 0;

    // Main cycle
    while (batches_done < max_batches_per_launch && ((rem[0]|rem[1]|rem[2]|rem[3]) != 0ull)) {
        if (warp_found_ready(d_found_flag, full_mask, lane)) { WARP_FLUSH_HASHES(); return; }

        bool rem_ge_batch = ge256_u64(rem, (uint64_t)batch);

        if (rem_ge_batch) {
            uint64_t subp[MAX_BATCH_SIZE/2][4];
            uint64_t acc[4], tmp[4];

#pragma unroll
            for (int j = 0; j < 4; ++j) acc[j] = g_Jx[j];
            ModSub256(acc, acc, x1);
#pragma unroll
            for (int j = 0; j < 4; ++j) subp[half - 1][j] = acc[j];

            for (int i = half - 1; i > 0; --i) {
#pragma unroll
                for (int j = 0; j < 4; ++j) tmp[j] = g_pGx[(size_t)i * 4 + j];
                ModSub256(tmp, tmp, x1);
                _ModMult(acc, acc, tmp);
#pragma unroll
                for (int j = 0; j < 4; ++j) subp[i - 1][j] = acc[j];
            }

            uint64_t d0[4];
#pragma unroll
            for (int j = 0; j < 4; ++j) d0[j] = g_pGx[0 * 4 + j];
            ModSub256(d0, d0, x1);

            uint64_t inverse[5];
#pragma unroll
            for (int j = 0; j < 4; ++j) inverse[j] = d0[j];
            _ModMult(inverse, subp[0]); 
            inverse[4] = 0ULL;
            _ModInv(inverse);

            for (int i = 0; i < half - 1; ++i) {
                uint64_t dx[4];
                _ModMult(dx, subp[i], inverse); // 1/(pGx[i]-x1)

                // +Pi
                {
                    uint64_t px_i[4], py_i[4];
#pragma unroll
                    for (int j = 0; j < 4; ++j) { px_i[j] = g_pGx[(size_t)i*4 + j]; py_i[j] = g_pGy[(size_t)i*4 + j]; }

                    uint64_t lam[4], x3[4], s[4];
                    ModSub256(s, py_i, y1);
                    _ModMult(lam, s, dx);

                    _ModSqr(x3, lam);
                    ModSub256(x3, x3, x1);
                    ModSub256(x3, x3, px_i);

                    ModSub256(s, x1, x3);
                    _ModMult(s, s, lam);
                    uint8_t parityY;
                    ModSub256isOdd(s, y1, &parityY);

                    uint8_t h20[20];
                    getHash160_33_from_limbs(parityY ? 0x03 : 0x02, x3, h20);
                    ++local_hashes; MAYBE_WARP_FLUSH();

                    bool pref = hash160_prefix_equals(h20, target_prefix);
                    if (__any_sync(full_mask, pref)) {
                        if (pref && hash160_matches_prefix_then_full(h20, c_target_hash160, target_prefix)) {
                            if (atomicCAS(d_found_flag, FOUND_NONE, FOUND_LOCK) == FOUND_NONE) {
                                d_found_result->threadId = (int)gid;
                                d_found_result->iter     = 0;

                                uint64_t fs[4];
#pragma unroll
                                for (int k=0;k<4;++k) fs[k]=base_scalar[k];
                                uint64_t carry=(uint64_t)(i+1);
#pragma unroll
                                for (int k=0;k<4 && carry;++k){ uint64_t old=fs[k]; fs[k]+=carry; carry=(fs[k]<old)?1:0; }
#pragma unroll
                                for (int k=0;k<4;++k) d_found_result->scalar[k]=fs[k];

#pragma unroll
                                for (int k=0;k<4;++k) d_found_result->Rx[k]=x3[k];

                                ModSub256(s, x1, x3);
                                _ModMult(s, s, lam);
                                uint64_t y3_full[4]; ModSub256(y3_full, s, y1);
#pragma unroll
                                for (int k=0;k<4;++k) d_found_result->Ry[k]=y3_full[k];

                                __threadfence_system();
                                atomicExch(d_found_flag, FOUND_READY);
                            }
                        }
                        __syncwarp(full_mask); WARP_FLUSH_HASHES(); return;
                    }
                }

                // -Pi
                {
                    uint64_t pxn[4], pyn[4];
#pragma unroll
                    for (int j=0;j<4;++j){ pxn[j]=g_pGx[(size_t)i*4 + j]; pyn[j]=g_pGy[(size_t)i*4 + j]; }
                    ModNeg256(pyn, pyn);

                    uint64_t lam[4], x3[4], s[4];
                    ModSub256(s, pyn, y1);
                    _ModMult(lam, s, dx);
                    _ModSqr(x3, lam);
                    ModSub256(x3, x3, x1);
                    ModSub256(x3, x3, pxn);
                    ModSub256(s, x1, x3);
                    _ModMult(s, s, lam);
                    uint8_t parityY;
                    ModSub256isOdd(s, y1, &parityY);

                    uint8_t h20[20];
                    getHash160_33_from_limbs(parityY ? 0x03 : 0x02, x3, h20);
                    ++local_hashes; MAYBE_WARP_FLUSH();

                    bool pref = hash160_prefix_equals(h20, target_prefix);
                    if (__any_sync(full_mask, pref)) {
                        if (pref && hash160_matches_prefix_then_full(h20, c_target_hash160, target_prefix)) {
                            if (atomicCAS(d_found_flag, FOUND_NONE, FOUND_LOCK) == FOUND_NONE) {
                                d_found_result->threadId = (int)gid;
                                d_found_result->iter     = 0;

                                uint64_t fs[4];
#pragma unroll
                                for (int k=0;k<4;++k) fs[k]=base_scalar[k];
                                uint64_t borrow=(uint64_t)(i+1);
#pragma unroll
                                for (int k=0;k<4 && borrow;++k){ uint64_t old=fs[k]; fs[k]=old-borrow; borrow=(old<borrow)?1:0; }
#pragma unroll
                                for (int k=0;k<4;++k) d_found_result->scalar[k]=fs[k];

#pragma unroll
                                for (int k=0;k<4;++k) d_found_result->Rx[k]=x3[k];

                                ModSub256(s, x1, x3);
                                _ModMult(s, s, lam);
                                uint64_t y3_full[4]; ModSub256(y3_full, s, y1);
#pragma unroll
                                for (int k=0;k<4;++k) d_found_result->Ry[k]=y3_full[k];

                                __threadfence_system();
                                atomicExch(d_found_flag, FOUND_READY);
                            }
                        }
                        __syncwarp(full_mask); WARP_FLUSH_HASHES(); return;
                    }
                }

#pragma unroll
                for (int j = 0; j < 4; ++j) tmp[j] = g_pGx[(size_t)i*4 + j];
                ModSub256(tmp, tmp, x1);
                _ModMult(inverse, tmp);
            }

            // Last-P_{half-1}
            {
                uint64_t dx_last[4];
                _ModMult(dx_last, subp[half - 1], inverse); // 1/(pGx[half-1]-x1)

                uint64_t pxn[4], pyn[4];
#pragma unroll
                for (int j=0;j<4;++j){ pxn[j]=g_pGx[(size_t)(half-1)*4 + j]; pyn[j]=g_pGy[(size_t)(half-1)*4 + j]; }
                ModNeg256(pyn, pyn);

                uint64_t lam[4], x3[4], s[4];
                ModSub256(s, pyn, y1);
                _ModMult(lam, s, dx_last);
                _ModSqr(x3, lam);
                ModSub256(x3, x3, x1);
                ModSub256(x3, x3, pxn);
                ModSub256(s, x1, x3);
                _ModMult(s, s, lam);
                uint8_t parityY;
                ModSub256isOdd(s, y1, &parityY);

                uint8_t h20[20];
                getHash160_33_from_limbs(parityY ? 0x03 : 0x02, x3, h20);
                ++local_hashes; MAYBE_WARP_FLUSH();

                bool pref = hash160_prefix_equals(h20, target_prefix);
                if (__any_sync(full_mask, pref)) {
                    if (pref && hash160_matches_prefix_then_full(h20, c_target_hash160, target_prefix)) {
                        if (atomicCAS(d_found_flag, FOUND_NONE, FOUND_LOCK) == FOUND_NONE) {
                            d_found_result->threadId = (int)gid;
                            d_found_result->iter     = 0;

                            uint64_t fs[4];
#pragma unroll
                            for (int k=0;k<4;++k) fs[k]=base_scalar[k];
                            uint64_t borrow=(uint64_t)(half);
#pragma unroll
                            for (int k=0;k<4 && borrow;++k){ uint64_t old=fs[k]; fs[k]=old-borrow; borrow=(old<borrow)?1:0; }
#pragma unroll
                            for (int k=0;k<4;++k) d_found_result->scalar[k]=fs[k];

#pragma unroll
                            for (int k=0;k<4;++k) d_found_result->Rx[k]=x3[k];

                            ModSub256(s, x1, x3);
                            _ModMult(s, s, lam);
                            uint64_t y3_full[4]; ModSub256(y3_full, s, y1);
#pragma unroll
                            for (int k=0;k<4;++k) d_found_result->Ry[k]=y3_full[k];

                            __threadfence_system();
                            atomicExch(d_found_flag, FOUND_READY);
                        }
                    }
                    __syncwarp(full_mask); WARP_FLUSH_HASHES(); return;
                }
            }

            uint64_t tmp2[4];
#pragma unroll
            for (int j = 0; j < 4; ++j) tmp2[j] = g_pGx[(size_t)(half-1)*4 + j];
            ModSub256(tmp2, tmp2, x1);
            _ModMult(inverse, tmp2);

            uint64_t dyJ[4], lamJ[4], xJ[4], sJ[4];
            ModSub256(dyJ, g_Jy, y1);
            _ModMult(lamJ, dyJ, inverse);

            _ModSqr(xJ, lamJ);
            ModSub256(xJ, xJ, x1);
            ModSub256(xJ, xJ, g_Jx);

            ModSub256(sJ, x1, xJ);
            _ModMult(sJ, sJ, lamJ);
            ModSub256(sJ, sJ, y1);

#pragma unroll
            for (int j=0;j<4;++j){ x1[j]=xJ[j]; y1[j]=sJ[j]; }

            {
                uint64_t carry = (uint64_t)batch;
#pragma unroll
                for (int k=0;k<4 && carry;++k){ uint64_t old=base_scalar[k]; base_scalar[k]+=carry; carry=(base_scalar[k]<old)?1:0; }
            }
            sub256_u64_inplace(rem, (uint64_t)batch);
            ++batches_done;
            continue;
        }

        //Tail batc
        uint64_t rem64 = to_u64_clamped(rem);
        uint32_t active = (rem[3] | rem[2] | rem[1]) ? (uint32_t)batch
                                                     : (uint32_t)((rem64 < (uint64_t)batch) ? rem64 : (uint64_t)batch);
        if (active == 0u) break;


        const uint32_t last_idx = active; 
        uint64_t dx[MAX_BATCH_SIZE + 1][4];
        uint64_t px[MAX_BATCH_SIZE]; 
        uint64_t py[MAX_BATCH_SIZE];

        uint64_t prod[MAX_BATCH_SIZE + 1][4];

        for (uint32_t k = 0; k < active; ++k) {
#pragma unroll
            for (int j=0;j<4;++j) dx[k][j] = g_pGx[(size_t)k*4 + j];
            ModSub256(dx[k], dx[k], x1);
        }

        uint64_t Jx[4], Jy[4];
#pragma unroll
        for (int j=0;j<4;++j) { Jx[j] = g_pGx[(size_t)(active-1)*4 + j]; Jy[j] = g_pGy[(size_t)(active-1)*4 + j]; }
#pragma unroll
        for (int j=0;j<4;++j) dx[last_idx][j] = Jx[j];
        ModSub256(dx[last_idx], dx[last_idx], x1);


#pragma unroll
        for (int j=0;j<4;++j) prod[0][j] = dx[0][j];
        for (uint32_t i = 1; i <= last_idx; ++i) {
            _ModMult(prod[i], prod[i-1], dx[i]);
        }

        uint64_t inv_total[5];
#pragma unroll
        for (int j=0;j<4;++j) inv_total[j] = prod[last_idx][j];
        inv_total[4] = 0ULL;
        _ModInv(inv_total);

        uint64_t inv_dx_k[4];
        uint64_t suffix[4] = { 1ull, 0ull, 0ull, 0ull }; 

        uint64_t inv_dx_last[4];
        if (last_idx > 0) {
            uint64_t t[4];
#pragma unroll
            for (int j=0;j<4;++j) t[j] = prod[last_idx - 1][j];
            _ModMult(inv_dx_last, inv_total, t);
        } else {
            // last_idx == 0 => inv_dx_last = inv_total
#pragma unroll
            for (int j=0;j<4;++j) inv_dx_last[j] = inv_total[j];
        }


        uint64_t acc_suffix[4] = { 1ull, 0ull, 0ull, 0ull };

        _ModMult(acc_suffix, acc_suffix, dx[last_idx]);

        for (int k = (int)active - 1; k >= 0; --k) {
            uint64_t left_prod[4];
            if (k > 0) {
#pragma unroll
                for (int j=0;j<4;++j) left_prod[j] = prod[k - 1][j];
            } else {
                // 1
                left_prod[0]=1ull; left_prod[1]=0ull; left_prod[2]=0ull; left_prod[3]=0ull;
            }
            uint64_t inv_dx_k_tmp[4];
            _ModMult(inv_dx_k_tmp, inv_total, left_prod);
            _ModMult(inv_dx_k, inv_dx_k_tmp, acc_suffix);

            {
                uint64_t px_i[4], py_i[4];
#pragma unroll
                for (int j = 0; j < 4; ++j) { px_i[j] = g_pGx[(size_t)k*4 + j]; py_i[j] = g_pGy[(size_t)k*4 + j]; }

                uint64_t lam[4], x3[4], s[4];
                ModSub256(s, py_i, y1);
                _ModMult(lam, s, inv_dx_k);

                _ModSqr(x3, lam);
                ModSub256(x3, x3, x1);
                ModSub256(x3, x3, px_i);

                ModSub256(s, x1, x3);
                _ModMult(s, s, lam);
                uint8_t parityY;
                ModSub256isOdd(s, y1, &parityY);

                uint8_t h20[20];
                getHash160_33_from_limbs(parityY ? 0x03 : 0x02, x3, h20);
                ++local_hashes; MAYBE_WARP_FLUSH();

                bool pref = hash160_prefix_equals(h20, target_prefix);
                if (__any_sync(full_mask, pref)) {
                    if (pref && hash160_matches_prefix_then_full(h20, c_target_hash160, target_prefix)) {
                        if (atomicCAS(d_found_flag, FOUND_NONE, FOUND_LOCK) == FOUND_NONE) {
                            d_found_result->threadId = (int)gid;
                            d_found_result->iter     = 0;

                            uint64_t fs[4];
#pragma unroll
                            for (int t=0;t<4;++t) fs[t]=base_scalar[t];
                            uint64_t addv = (uint64_t)(k + 1);
#pragma unroll
                            for (int t=0;t<4 && addv;++t){ uint64_t old=fs[t]; uint64_t sum=old+addv; fs[t]=sum; addv=(sum<old)?1ull:0ull; }
#pragma unroll
                            for (int t=0;t<4;++t) d_found_result->scalar[t]=fs[t];

#pragma unroll
                            for (int t=0;t<4;++t) d_found_result->Rx[t]=x3[t];
                            uint64_t y3_full[4]; ModSub256(y3_full, s, y1);
#pragma unroll
                            for (int t=0;t<4;++t) d_found_result->Ry[t]=y3_full[t];

                            __threadfence_system();
                            atomicExch(d_found_flag, FOUND_READY);
                        }
                    }
                    __syncwarp(full_mask); WARP_FLUSH_HASHES(); return;
                }
            }

            _ModMult(acc_suffix, acc_suffix, dx[k]);
        }

        {
            uint64_t dyJ[4], lamJ[4], xJ2[4], sJ2[4];
            ModSub256(dyJ, Jy, y1);
            _ModMult(lamJ, dyJ, inv_dx_last);

            _ModSqr(xJ2, lamJ);
            ModSub256(xJ2, xJ2, x1);
            ModSub256(xJ2, xJ2, Jx);

            ModSub256(sJ2, x1, xJ2);
            _ModMult(sJ2, sJ2, lamJ);
            ModSub256(sJ2, sJ2, y1);

#pragma unroll
            for (int j=0;j<4;++j){ x1[j]=xJ2[j]; y1[j]=sJ2[j]; }
        }

        {
            uint64_t carry = (uint64_t)active;
#pragma unroll
            for (int k=0;k<4 && carry;++k){ uint64_t old=base_scalar[k]; base_scalar[k]+=carry; carry=(base_scalar[k]<old)?1:0; }
        }
        sub256_u64_inplace(rem, (uint64_t)active);
        ++batches_done;
    }

#pragma unroll
    for (int i = 0; i < 4; ++i) {
        Rx[gid*4+i] = x1[i];
        Ry[gid*4+i] = y1[i];
        counts256[gid*4+i] = rem[i];
        start_scalars[gid*4+i] = base_scalar[i];
    }

    if ((rem[0] | rem[1] | rem[2] | rem[3]) != 0ull) {
        atomicAdd(d_any_left, 1u);
    }

    WARP_FLUSH_HASHES();

    #undef MAYBE_WARP_FLUSH
    #undef WARP_FLUSH_HASHES
    #undef FLUSH_THRESHOLD
}

static inline bool is_zero_256_host(const uint64_t a[4]) {
    return (a[0]|a[1]|a[2]|a[3]) == 0ull;
}

static void divmod_256_by_u64_safe(const uint64_t a[4], uint64_t d,
                                   uint64_t q[4], uint64_t &r)
{
    unsigned __int128 rem = 0;
    uint64_t t[4] = { a[3], a[2], a[1], a[0] }; 

    uint64_t q_be[4];
    for (int i = 0; i < 4; ++i) {
        unsigned __int128 cur = (rem << 64) | t[i];
        uint64_t qword = (uint64_t)(cur / d);
        rem = (cur % d);
        q_be[i] = qword;
    }
    q[0] = q_be[3];
    q[1] = q_be[2];
    q[2] = q_be[1];
    q[3] = q_be[0];
    r = (uint64_t)rem;
}

int main(int argc, char** argv) {
    std::string target_hash_hex, range_hex;
    std::string address_b58;
    bool grid_provided = false;
    uint32_t runtime_points_batch_size = 128;
    uint32_t runtime_batches_per_sm    = 8;   
    uint32_t slices_per_launch         = 64; 

    auto parse_grid = [](const std::string& s, uint32_t& a_out, uint32_t& b_out)->bool {
        size_t comma = s.find(',');
        if (comma == std::string::npos) return false;
        auto trim = [](std::string& z){
            size_t p1 = z.find_first_not_of(" \t");
            size_t p2 = z.find_last_not_of(" \t");
            if (p1 == std::string::npos) { z.clear(); return; }
            z = z.substr(p1, p2 - p1 + 1);
        };
        std::string a_str = s.substr(0, comma);
        std::string b_str = s.substr(comma + 1);
        trim(a_str); trim(b_str);
        if (a_str.empty() || b_str.empty()) return false;
        char* endp = nullptr;
        unsigned long aa = std::strtoul(a_str.c_str(), &endp, 10);
        if (*endp != '\0') return false;
        endp = nullptr;
        unsigned long bb = std::strtoul(b_str.c_str(), &endp, 10);
        if (*endp != '\0') return false;
        if (aa == 0ul || bb == 0ul) return false;
        if (aa > (1ul<<20) || bb > (1ul<<20)) return false;
        a_out = (uint32_t)aa;
        b_out = (uint32_t)bb;
        return true;
    };

    for (int i = 1; i < argc; ++i) {
        std::string arg = argv[i];
        if      (arg == "--target-hash160" && i + 1 < argc) target_hash_hex = argv[++i];
        else if (arg == "--address"        && i + 1 < argc) address_b58     = argv[++i];
        else if (arg == "--range"          && i + 1 < argc) range_hex       = argv[++i];
        else if (arg == "--grid"           && i + 1 < argc) {
            uint32_t a=0,b=0;
            if (!parse_grid(argv[++i], a, b)) {
                std::cerr << "Error: --grid expects \"A,B\" (positive integers).\n";
                return EXIT_FAILURE;
            }
            runtime_points_batch_size = a;
            runtime_batches_per_sm    = b;
            grid_provided = true;
        }
        else if (arg == "--slices" && i + 1 < argc) {
            char* endp=nullptr;
            unsigned long v = std::strtoul(argv[++i], &endp, 10);
            if (*endp != '\0' || v == 0ul || v > (1ul<<20)) {
                std::cerr << "Error: --slices must be in 1.." << (1u<<20) << "\n";
                return EXIT_FAILURE;
            }
            slices_per_launch = (uint32_t)v;
        }
    }

    if (range_hex.empty() || (target_hash_hex.empty() && address_b58.empty())) {
        std::cerr << "Usage: " << argv[0]
                  << " --range <start_hex>:<end_hex> (--address <base58> | --target-hash160 <hash160_hex>) [--grid A,B] [--slices N]\n";
        return EXIT_FAILURE;
    }
    if (!target_hash_hex.empty() && !address_b58.empty()) {
        std::cerr << "Error: provide either --address or --target-hash160, not both.\n";
        return EXIT_FAILURE;
    }

    size_t colon_pos = range_hex.find(':');
    if (colon_pos == std::string::npos) { std::cerr << "Error: range format must be start:end\n"; return EXIT_FAILURE; }
    std::string start_hex = range_hex.substr(0, colon_pos);
    std::string end_hex   = range_hex.substr(colon_pos + 1);

    uint64_t range_start[4]{0}, range_end[4]{0};
    if (!hexToLE64(start_hex, range_start) || !hexToLE64(end_hex, range_end)) {
        std::cerr << "Error: invalid range hex\n"; return EXIT_FAILURE;
    }

    uint8_t target_hash160[20];
    if (!address_b58.empty()) {
        if (!decode_p2pkh_address(address_b58, target_hash160)) {
            std::cerr << "Error: invalid P2PKH address (Base58Check failed or wrong version)\n";
            return EXIT_FAILURE;
        }
    } else {
        if (!hexToHash160(target_hash_hex, target_hash160)) {
            std::cerr << "Error: Invalid target hash160 hex\n"; return EXIT_FAILURE;
        }
    }

    auto is_pow2 = [](uint32_t v)->bool { return v && ((v & (v-1)) == 0); };
    if (!is_pow2(runtime_points_batch_size) || (runtime_points_batch_size & 1u)) {
        std::cerr << "Error: batch size must be even and a power of two.\n";
        return EXIT_FAILURE;
    }
    if (runtime_points_batch_size > MAX_BATCH_SIZE) {
        std::cerr << "Error: batch size must be <= " << MAX_BATCH_SIZE << ".\n";
        return EXIT_FAILURE;
    }
    if (slices_per_launch == 0) {
        std::cerr << "Error: slices_per_launch must be > 0\n";
        return EXIT_FAILURE;
    }

    // range_len = end - start + 1
    uint64_t range_len[4];
    sub256(range_end, range_start, range_len);
    add256_u64(range_len, 1ull, range_len);

    auto is_zero_256 = [](const uint64_t a[4])->bool {
        return (a[0]|a[1]|a[2]|a[3]) == 0ull;
    };
    auto is_power_of_two_256 = [&](const uint64_t a[4])->bool {
        if (is_zero_256(a)) return false;
        uint64_t am1[4];
        uint64_t borrow = 1ull;
        for (int i=0;i<4;++i) {
            uint64_t v = a[i] - borrow;
            borrow = (a[i] < borrow) ? 1ull : 0ull;
            am1[i] = v;
            if (borrow == 0ull && i+1<4) { for (int k=i+1;k<4;++k) am1[k] = a[k]; break; }
        }
        uint64_t andv0 = a[0] & am1[0];
        uint64_t andv1 = a[1] & am1[1];
        uint64_t andv2 = a[2] & am1[2];
        uint64_t andv3 = a[3] & am1[3];
        return (andv0|andv1|andv2|andv3) == 0ull;
    };
    if (!is_power_of_two_256(range_len)) {
        std::cerr << "Error: range length (end - start + 1) must be a power of two.\n";
        return EXIT_FAILURE;
    }
    uint64_t len_minus1[4];
    {
        uint64_t borrow = 1ull;
        for (int i=0;i<4;++i) {
            uint64_t v = range_len[i] - borrow;
            borrow = (range_len[i] < borrow) ? 1ull : 0ull;
            len_minus1[i] = v;
            if (borrow == 0ull && i+1<4) { for (int k=i+1;k<4;++k) len_minus1[k] = range_len[k]; break; }
        }
    }
    {
        uint64_t and0 = range_start[0] & len_minus1[0];
        uint64_t and1 = range_start[1] & len_minus1[1];
        uint64_t and2 = range_start[2] & len_minus1[2];
        uint64_t and3 = range_start[3] & len_minus1[3];
        if ((and0|and1|and2|and3) != 0ull) {
            std::cerr << "Error: start must be aligned to the range length (power-of-two aligned).\n";
            return EXIT_FAILURE;
        }
    }

    int device = 0;
    hipDeviceProp_t prop{};
    hipError_t err = hipGetDevice(&device);
    if (err != hipSuccess) { std::cerr << "hipGetDevice error\n"; return EXIT_FAILURE; }
    if (hipGetDeviceProperties(&prop, device) != hipSuccess) { std::cerr << "hipGetDeviceProperties error\n"; return EXIT_FAILURE; }

    size_t stackSize = 64 * 1024;
    hipDeviceSetLimit(hipLimitStackSize, stackSize);

    int threadsPerBlock = 256;
    if (threadsPerBlock > (int)prop.maxThreadsPerBlock) threadsPerBlock = prop.maxThreadsPerBlock;
    if (threadsPerBlock < 32) threadsPerBlock = 32;

    const uint64_t bytesPerThread = 2 * 4 * sizeof(uint64_t);
    size_t totalGlobalMem = prop.totalGlobalMem;
    const uint64_t reserveBytes = 64ull * 1024 * 1024;
    uint64_t usableMem = (totalGlobalMem > reserveBytes) ? (totalGlobalMem - reserveBytes) : (totalGlobalMem / 2);
    uint64_t maxThreadsByMem = usableMem / bytesPerThread;

    uint64_t q_div_batch[4], r_div_batch = 0;
    divmod_256_by_u64_safe(range_len, (uint64_t)runtime_points_batch_size, q_div_batch, r_div_batch);
    if (r_div_batch != 0ull) {
        std::cerr << "Error: range length must be divisible by batch size (" << runtime_points_batch_size << ").\n";
        return EXIT_FAILURE;
    }
    bool q_fits_u64 = (q_div_batch[3] | q_div_batch[2] | q_div_batch[1]) == 0ull;
    uint64_t q_u64  = q_fits_u64 ? q_div_batch[0] : UINT64_MAX;

    uint64_t userUpper = (uint64_t)prop.multiProcessorCount * (uint64_t)runtime_batches_per_sm * (uint64_t)threadsPerBlock;
    if (userUpper == 0ull) userUpper = UINT64_MAX;

    auto pick_threads_total = [&](uint64_t upper)->uint64_t {
        if (upper < (uint64_t)threadsPerBlock) return 0ull;
        uint64_t t = upper - (upper % (uint64_t)threadsPerBlock);
        if (!q_fits_u64) return t;
        uint64_t q = q_u64;
        while (t >= (uint64_t)threadsPerBlock) {
            if ((q % t) == 0ull) return t;
            t -= (uint64_t)threadsPerBlock;
        }
        return 0ull;
    };

    uint64_t upper = maxThreadsByMem;
    if (q_fits_u64 && q_u64 < upper) upper = q_u64;
    if (userUpper   < upper)         upper = userUpper;

    uint64_t threadsTotal = pick_threads_total(upper);
    if (threadsTotal == 0ull) {
        std::cerr << "Error: failed to pick threadsTotal satisfying divisibility.\n";
        return EXIT_FAILURE;
    }
    int blocks = (int)(threadsTotal / (uint64_t)threadsPerBlock);

    uint64_t q256[4]; uint64_t r_u64 = 0;
    divmod_256_by_u64_safe(range_len, threadsTotal, q256, r_u64);
    if (r_u64 != 0ull) {
        std::cerr << "Internal error: range_len not divisible by threadsTotal after alignment.\n";
        return EXIT_FAILURE;
    }
    {
        uint64_t qq[4], rr = 0;
        divmod_256_by_u64_safe(q256, (uint64_t)runtime_points_batch_size, qq, rr);
        if (rr != 0ull) {
            std::cerr << "Internal error: per-thread count is not a multiple of batch size.\n";
            return EXIT_FAILURE;
        }
    }

    uint64_t* h_counts256     = new uint64_t[threadsTotal * 4];
    uint64_t* h_start_scalars = new uint64_t[threadsTotal * 4];

    for (uint64_t i = 0; i < threadsTotal; ++i) {
        h_counts256[i*4+0] = q256[0];
        h_counts256[i*4+1] = q256[1];
        h_counts256[i*4+2] = q256[2];
        h_counts256[i*4+3] = q256[3];
    }
    {
        uint64_t cur[4] = { range_start[0], range_start[1], range_start[2], range_start[3] };
        for (uint64_t i = 0; i < threadsTotal; ++i) {
            h_start_scalars[i*4+0] = cur[0];
            h_start_scalars[i*4+1] = cur[1];
            h_start_scalars[i*4+2] = cur[2];
            h_start_scalars[i*4+3] = cur[3];
            uint64_t next[4];
            add256(cur, &h_counts256[i*4], next);
            cur[0]=next[0]; cur[1]=next[1]; cur[2]=next[2]; cur[3]=next[3];
        }
    }

    {
        uint32_t prefix_le = (uint32_t)target_hash160[0]
                           | ((uint32_t)target_hash160[1] << 8)
                           | ((uint32_t)target_hash160[2] << 16)
                           | ((uint32_t)target_hash160[3] << 24);
        hipMemcpyToSymbol(HIP_SYMBOL(c_target_prefix), &prefix_le, sizeof(prefix_le));
        hipMemcpyToSymbol(HIP_SYMBOL(c_target_hash160), target_hash160, 20);
    }

    uint64_t *d_start_scalars=nullptr, *d_Px=nullptr, *d_Py=nullptr, *d_Rx=nullptr, *d_Ry=nullptr, *d_counts256=nullptr;
    int *d_found_flag=nullptr;
    FoundResult *d_found_result=nullptr;
    unsigned long long *d_hashes_accum=nullptr;
    unsigned int *d_any_left=nullptr;

    hipMalloc(&d_start_scalars, threadsTotal * 4 * sizeof(uint64_t));
    hipMalloc(&d_Px, threadsTotal * 4 * sizeof(uint64_t));
    hipMalloc(&d_Py, threadsTotal * 4 * sizeof(uint64_t));
    hipMalloc(&d_Rx, threadsTotal * 4 * sizeof(uint64_t));
    hipMalloc(&d_Ry, threadsTotal * 4 * sizeof(uint64_t));
    hipMalloc(&d_counts256, threadsTotal * 4 * sizeof(uint64_t));
    hipMalloc(&d_found_flag, sizeof(int));
    hipMalloc(&d_found_result, sizeof(FoundResult));
    hipMalloc(&d_hashes_accum, sizeof(unsigned long long));
    hipMalloc(&d_any_left, sizeof(unsigned int));

    hipMemcpy(d_start_scalars, h_start_scalars, threadsTotal * 4 * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_counts256,     h_counts256,     threadsTotal * 4 * sizeof(uint64_t), hipMemcpyHostToDevice);
    {
        int zero = FOUND_NONE;
        hipMemcpy(d_found_flag, &zero, sizeof(int), hipMemcpyHostToDevice);
        unsigned long long zero64 = 0ull;
        hipMemcpy(d_hashes_accum, &zero64, sizeof(unsigned long long), hipMemcpyHostToDevice);
    }

    {
        int blocks_scal = (int)((threadsTotal + threadsPerBlock - 1) / threadsPerBlock);
        scalarMulKernelBase<<<blocks_scal, threadsPerBlock>>>(d_start_scalars, d_Px, d_Py, (int)threadsTotal);
        hipDeviceSynchronize();
    }

    {
        const uint32_t B = runtime_points_batch_size;

        uint64_t *d_pGx=nullptr, *d_pGy=nullptr;
        hipMalloc(&d_pGx, (size_t)B * 4 * sizeof(uint64_t));
        hipMalloc(&d_pGy, (size_t)B * 4 * sizeof(uint64_t));

        uint64_t* h_scal = new uint64_t[(size_t)B * 4];
        std::memset(h_scal, 0, (size_t)B * 4 * sizeof(uint64_t));
        for (uint32_t k = 0; k < B; ++k) h_scal[(size_t)k*4 + 0] = (uint64_t)(k + 1);

        uint64_t *d_pG_scalars=nullptr;
        hipMalloc(&d_pG_scalars, (size_t)B * 4 * sizeof(uint64_t));
        hipMemcpy(d_pG_scalars, h_scal, (size_t)B * 4 * sizeof(uint64_t), hipMemcpyHostToDevice);

        int blocks_scal = (int)((B + threadsPerBlock - 1) / threadsPerBlock);
        scalarMulKernelBase<<<blocks_scal, threadsPerBlock>>>(d_pG_scalars, d_pGx, d_pGy, (int)B);
        hipDeviceSynchronize();

        hipMemcpyToSymbol(HIP_SYMBOL(g_pGx), d_pGx, (size_t)B * 4 * sizeof(uint64_t), 0, hipMemcpyDeviceToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(g_pGy), d_pGy, (size_t)B * 4 * sizeof(uint64_t), 0, hipMemcpyDeviceToDevice);

        // J = B*G
        hipMemcpyToSymbol(HIP_SYMBOL(g_Jx), d_pGx + (size_t)(B - 1) * 4, 4 * sizeof(uint64_t), 0, hipMemcpyDeviceToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(g_Jy), d_pGy + (size_t)(B - 1) * 4, 4 * sizeof(uint64_t), 0, hipMemcpyDeviceToDevice);

        hipFree(d_pG_scalars);
        delete[] h_scal;
        hipFree(d_pGx);
        hipFree(d_pGy);
    }

    size_t freeB=0,totalB=0;
    hipMemGetInfo(&freeB,&totalB);
    size_t usedB = totalB - freeB;
    double util = totalB ? (double)usedB * 100.0 / (double)totalB : 0.0;

    std::cout << "======== PrePhase: GPU Information ====================\n";
    std::cout << std::left << std::setw(20) << "Device"            << " : " << prop.name << " (compute " << prop.major << "." << prop.minor << ")\n";
    std::cout << std::left << std::setw(20) << "SM"                << " : " << prop.multiProcessorCount << "\n";
    std::cout << std::left << std::setw(20) << "ThreadsPerBlock"   << " : " << threadsPerBlock << "\n";
    std::cout << std::left << std::setw(20) << "Blocks"            << " : " << blocks << "\n";
    std::cout << std::left << std::setw(20) << "Points batch size" << " : " << runtime_points_batch_size << "\n";
    std::cout << std::left << std::setw(20) << "Batches/SM"        << " : " << runtime_batches_per_sm << "\n";
    std::cout << std::left << std::setw(20) << "Batches/launch"    << " : " << slices_per_launch << " (per thread)\n";
    std::cout << std::left << std::setw(20) << "Memory utilization"<< " : "
              << std::fixed << std::setprecision(1) << util << "% ("
              << human_bytes((double)usedB) << " / " << human_bytes((double)totalB) << ")\n";
    std::cout << "------------------------------------------------------- \n";
    std::cout << std::left << std::setw(20) << "Total threads"     << " : " << threadsTotal << "\n\n";

    std::cout << "======== Phase-1: BruteForce (sliced) =================\n";

    hipStream_t streamKernel;
    hipStreamCreateWithFlags(&streamKernel, hipStreamNonBlocking);

    auto t0 = std::chrono::high_resolution_clock::now();
    auto tLast = t0;
    unsigned long long lastHashes = 0ull;

    size_t sharedBytes = 0;

    bool first_launch = true;
    bool stop_all = false;

    while (!stop_all) {
        unsigned int zeroU = 0u;
        hipMemcpyAsync(d_any_left, &zeroU, sizeof(unsigned int), hipMemcpyHostToDevice, streamKernel);

        kernel_point_add_and_check_sliced<<<blocks, threadsPerBlock, sharedBytes, streamKernel>>>(
            d_Px, d_Py, d_Rx, d_Ry,
            d_start_scalars,
            d_counts256,
            threadsTotal,
            runtime_points_batch_size,
            slices_per_launch,
            first_launch ? 1 : 0,
            d_found_flag, d_found_result,
            d_hashes_accum,
            d_any_left
        );
        hipGetLastError();

        while (true) {
            auto now = std::chrono::high_resolution_clock::now();
            double dt = std::chrono::duration<double>(now - tLast).count();
            if (dt >= 1.0) {
                unsigned long long h_hashes = 0ull;
                hipMemcpy(&h_hashes, d_hashes_accum, sizeof(unsigned long long), hipMemcpyDeviceToHost);
                double delta = (double)(h_hashes - lastHashes);
                double mkeys = delta / (dt * 1e6);
                double elapsed = std::chrono::duration<double>(now - t0).count();
                long double total_keys_ld = ld_from_u256(range_len);
                long double prog = total_keys_ld > 0.0L ? ((long double)h_hashes / total_keys_ld) * 100.0L : 0.0L;
                if (prog > 100.0L) prog = 100.0L;
                std::cout << "\rTime: " << std::fixed << std::setprecision(1) << elapsed
                          << " s | Speed: " << std::fixed << std::setprecision(1) << mkeys
                          << " Mkeys/s | Count: " << h_hashes
                          << " | Progress: " << std::fixed << std::setprecision(2) << (double)prog << " %";
                std::cout.flush();
                lastHashes = h_hashes;
                tLast = now;
            }

            int host_found = 0;
            hipMemcpy(&host_found, d_found_flag, sizeof(int), hipMemcpyDeviceToHost);
            if (host_found == FOUND_READY) { stop_all = true; break; }

            hipError_t qs = hipStreamQuery(streamKernel);
            if (qs == hipSuccess) break;
            else if (qs != hipErrorNotReady) { hipGetLastError(); stop_all = true; break; }

            std::this_thread::sleep_for(std::chrono::milliseconds(10));
        }

        hipStreamSynchronize(streamKernel);
        std::cout.flush();

        if (stop_all) break;

        unsigned int h_any = 0u;
        hipMemcpy(&h_any, d_any_left, sizeof(unsigned int), hipMemcpyDeviceToHost);

        std::swap(d_Px, d_Rx);
        std::swap(d_Py, d_Ry);

        first_launch = false;

        if (h_any == 0u) {
            break;
        }
    }

    hipDeviceSynchronize();
    std::cout << "\n";

    int h_found_flag = 0;
    hipMemcpy(&h_found_flag, d_found_flag, sizeof(int), hipMemcpyDeviceToHost);
    if (h_found_flag == FOUND_READY) {
        FoundResult host_result{};
        hipMemcpy(&host_result, d_found_result, sizeof(FoundResult), hipMemcpyDeviceToHost);
        std::cout << "\n";
        std::cout << "======== FOUND MATCH! =================================\n";
        std::cout << "Private Key   : " << formatHex256(host_result.scalar) << "\n";
        std::cout << "Public Key    : " << formatCompressedPubHex(host_result.Rx, host_result.Ry) << "\n";
    }

    hipFree(d_start_scalars);
    hipFree(d_Px);
    hipFree(d_Py);
    hipFree(d_Rx);
    hipFree(d_Ry);
    hipFree(d_counts256);
    hipFree(d_found_flag);
    hipFree(d_found_result);
    hipFree(d_hashes_accum);
    hipFree(d_any_left);
    hipStreamDestroy(streamKernel);

    delete[] h_start_scalars;
    delete[] h_counts256;

    return 0;
}
