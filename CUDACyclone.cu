#include "hip/hip_runtime.h"
// CUDACyclone.cu — фикс первого батча: anchor + (B-1), затем всегда по B ключей.
// Предрасчёт 1..B·G выполняется один раз и хранится в девайс-памяти.
// Полный батч: ровно B проверок (± вокруг якоря, включая +P(half-1)).
// Хвост: только +Pk, k=0..active-1, ровно active проверок.
// Перенос якоря: на B·G (для полного) или на active·G (для хвоста).

#include <hip/hip_runtime.h>
#include <>
#include <cstdint>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <iostream>
#include <iomanip>
#include <sstream>
#include <string>
#include <thread>
#include <chrono>

#include "CUDAMath.h"         // ModSub256/_ModMult/_ModSqr/_ModInv/...
#include "CUDAHash.cuh"       // getHash160_33_from_limbs, c_target_prefix/c_target_hash160
#include "CUDAUtils.h"        // eq/ge/sub etc. (не дублируем)
#include "sha256.h"        // eq/ge/sub etc. (не дублируем)
#include "CUDAStructures.h"   // FoundResult, FOUND_*

#ifndef MAX_BATCH_SIZE
#define MAX_BATCH_SIZE 1024
#endif
#ifndef WARP_SIZE
#define WARP_SIZE 32
#endif

// ===== DEBUG: проверка количества хешей в батче/хвосте (gid==0) =====
#define DEBUG_BATCH_CHECK 0

// ===== Глобальные device-символы для предрасчитанных точек =====
__device__ uint64_t g_pGx[MAX_BATCH_SIZE * 4];
__device__ uint64_t g_pGy[MAX_BATCH_SIZE * 4];

// Утилиты
__device__ __forceinline__ int load_found_flag_relaxed(const int* p) {
    return *((const volatile int*)p);
}
__device__ __forceinline__ bool warp_found_ready(const int* __restrict__ d_found_flag,
                                                 unsigned full_mask,
                                                 unsigned lane)
{
    int f = 0;
    if (lane == 0) f = load_found_flag_relaxed(d_found_flag);
    f = __shfl_sync(full_mask, f, 0);
    return f == FOUND_READY;
}

// Объявление вашего ядра умножения скалярного (уже реализовано в проекте)
__global__ void scalarMulKernelBase(const uint64_t* scalars_in, uint64_t* outX, uint64_t* outY, int N);

// Деление 256 на u64 (безопасно для нашего представления)
static void divmod_256_by_u64_safe(const uint64_t a[4], uint64_t d,
                                   uint64_t q[4], uint64_t &r)
{
    unsigned __int128 rem = 0;
    uint64_t t[4] = { a[3], a[2], a[1], a[0] };

    uint64_t q_be[4];
    for (int i = 0; i < 4; ++i) {
        unsigned __int128 cur = (rem << 64) | t[i];
        uint64_t qword = (uint64_t)(cur / d);
        rem = (cur % d);
        q_be[i] = qword;
    }
    q[0] = q_be[3];
    q[1] = q_be[2];
    q[2] = q_be[1];
    q[3] = q_be[0];
    r = (uint64_t)rem;
}

// ===== Ядро: 1 запуск = 1 батч (± для полного, +Pk для хвоста) =====
// ВАЖНО: если do_initial_anchor_check != 0, то:
//  - проверяем якорь (1 ключ),
//  - затем считаем актив = min(B-1, rem) в хвостовой ветке (+Pk),
//  - итого за первый запуск ровно B ключей.
// Далее запуск всегда считает ровно B ключей (полный батч).
__launch_bounds__(256, 2)
__global__ void kernel_one_batch_pm_both(
    const uint64_t* __restrict__ Px,
    const uint64_t* __restrict__ Py,
    uint64_t* __restrict__ Rx,
    uint64_t* __restrict__ Ry,
    uint64_t* __restrict__ start_scalars,
    uint64_t* __restrict__ counts256,
    uint64_t threadsTotal,
    uint32_t batch_size,
    int do_initial_anchor_check,
    int* __restrict__ d_found_flag,
    FoundResult* __restrict__ d_found_result,
    unsigned long long* __restrict__ hashes_accum
)
{
    const uint64_t gid = (uint64_t)blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= threadsTotal) return;

    const unsigned lane      = (unsigned)(threadIdx.x & (WARP_SIZE - 1));
    const unsigned full_mask = 0xFFFFFFFFu;

    if (warp_found_ready(d_found_flag, full_mask, lane)) return;

    const int B = (int)batch_size;
    if (B <= 0 || (B & 1) || B > MAX_BATCH_SIZE) return;
    const int half = B >> 1;

    // состояние потока
    uint64_t X[4], Y[4], S[4], rem[4];
#pragma unroll
    for (int i=0;i<4;++i) {
        uint64_t idx = gid*4 + i;
        X[i]   = Px[idx];
        Y[i]   = Py[idx];
        S[i]   = start_scalars[idx];
        rem[i] = counts256[idx];
    }

    if ((rem[0]|rem[1]|rem[2]|rem[3]) == 0ull) {
#pragma unroll
        for (int i=0;i<4;++i) {
            Rx[gid*4+i] = X[i];
            Ry[gid*4+i] = Y[i];
        }
        return;
    }

    unsigned int local_hashes = 0;

#if DEBUG_BATCH_CHECK
    unsigned tested = 0;
#endif

    // 1) ЯКОРЬ (только при первом запуске): съедаем 1 ключ
    bool initial = (do_initial_anchor_check != 0);
    if (initial) {
        // hash(anchor)
        uint8_t h20[20];
        uint8_t odd = (uint8_t)(Y[0] & 1ULL) ? 0x03 : 0x02;
        getHash160_33_from_limbs(odd, X, h20);
        ++local_hashes;
#if DEBUG_BATCH_CHECK
        ++tested;
#endif
        bool pref = (*(const uint32_t*)h20 == c_target_prefix);
        if (__any_sync(full_mask, pref)) {
            if (pref && hash160_matches_prefix_then_full(h20, c_target_hash160, c_target_prefix)) {
                if (atomicCAS(d_found_flag, FOUND_NONE, FOUND_LOCK) == FOUND_NONE) {
#pragma unroll
                    for (int i=0;i<4;++i) { d_found_result->scalar[i]=S[i]; d_found_result->Rx[i]=X[i]; d_found_result->Ry[i]=Y[i]; }
                    d_found_result->threadId = (int)gid;
                    d_found_result->iter = 0;
                    __threadfence_system();
                    atomicExch(d_found_flag, FOUND_READY);
                }
            }
            atomicAdd(hashes_accum, (unsigned long long)local_hashes);
            return;
        }
        // rem -= 1; S += 1
        sub256_u64_inplace(rem, 1ull);
        if ((rem[0]|rem[1]|rem[2]|rem[3]) == 0ull) {
#pragma unroll
            for (int i=0;i<4;++i){ Rx[gid*4+i]=X[i]; Ry[gid*4+i]=Y[i]; counts256[gid*4+i]=rem[i]; start_scalars[gid*4+i]=S[i]; }
            atomicAdd(hashes_accum, (unsigned long long)local_hashes);
            return;
        }
        inc256_device(S, 1ull);
    }

    // 2) Определим, сколько ключей надо проверить в ЭТОМ запуске
    //    - если initial: осталось вместить только (B-1)
    //    - иначе: целый батч B
    uint32_t effective_B = initial ? (uint32_t)(B - 1) : (uint32_t)B;

    // Для корректного покрытия делаем: если effective_B == B и rem >= B → полный батч (±),
    // иначе — хвост (active = min(effective_B, rem)).
    bool can_full_batch = (!initial) && ge256_u64(rem, (uint64_t)B);

    if (can_full_batch) {
        // ===== ПОЛНЫЙ БАТЧ: ровно B точек (± вокруг якоря, включая +P(half-1)) =====
        uint64_t subp[MAX_BATCH_SIZE/2][4];
        uint64_t acc[4], tmp[4];

        // subp построение: acc = (pGx[half-1] - X)
#pragma unroll
        for (int j=0;j<4;++j) acc[j] = g_pGx[(size_t)(half-1)*4 + j];
        ModSub256(acc, acc, X);
#pragma unroll
        for (int j=0;j<4;++j) subp[half-1][j] = acc[j];

        for (int i = half-1; i > 0; --i) {
#pragma unroll
            for (int j=0;j<4;++j) tmp[j] = g_pGx[(size_t)i*4 + j];
            ModSub256(tmp, tmp, X);
            _ModMult(acc, acc, tmp);
#pragma unroll
            for (int j=0;j<4;++j) subp[i-1][j] = acc[j];
        }

        // inverse = 1 / ∏_{k=0..half-1}(pGx[k]-X)
        uint64_t inverse[5];
#pragma unroll
        for (int j=0;j<4;++j) inverse[j] = g_pGx[0*4 + j];
        ModSub256(inverse, inverse, X);
        _ModMult(inverse, subp[0]);
        inverse[4] = 0ull; _ModInv(inverse);

        // i=0..half-2 → +Pi и -Pi (2*(half-1) точек)
        for (int i=0; i<half-1; ++i) {
            uint64_t dx[4]; _ModMult(dx, subp[i], inverse);

            // +Pi
            {
                uint64_t px_i[4], py_i[4];
#pragma unroll
                for (int j=0;j<4;++j) { px_i[j]=g_pGx[(size_t)i*4 + j]; py_i[j]=g_pGy[(size_t)i*4 + j]; }
                uint64_t dy[4], lam[4], x3[4], s[4];
                ModSub256(dy, py_i, Y);
                _ModMult(lam, dy, dx);
                _ModSqr(x3, lam);
                ModSub256(x3, x3, X);
                ModSub256(x3, x3, px_i);
                ModSub256(s, X, x3);
                _ModMult(s, s, lam);
                uint8_t odd; ModSub256isOdd(s, Y, &odd);

                uint8_t h20[20]; getHash160_33_from_limbs(odd?0x03:0x02, x3, h20);
                ++local_hashes;
#if DEBUG_BATCH_CHECK
                ++tested;
#endif
                bool pref = (*(const uint32_t*)h20 == c_target_prefix);
                if (__any_sync(full_mask, pref)) {
                    if (pref && hash160_matches_prefix_then_full(h20, c_target_hash160, c_target_prefix)) {
                        if (atomicCAS(d_found_flag, FOUND_NONE, FOUND_LOCK) == FOUND_NONE) {
                            uint64_t fs[4];
#pragma unroll
                            for (int k=0;k<4;++k) fs[k]=S[k];
                            uint64_t addv=(uint64_t)(i+1);
#pragma unroll
                            for (int k=0;k<4 && addv;++k){ uint64_t old=fs[k]; fs[k]=old+addv; addv=(fs[k]<old)?1ull:0ull; }
#pragma unroll
                            for (int k=0;k<4;++k){ d_found_result->scalar[k]=fs[k]; d_found_result->Rx[k]=x3[k]; }
                            uint64_t y3_full[4]; ModSub256(y3_full, s, Y);
#pragma unroll
                            for (int k=0;k<4;++k) d_found_result->Ry[k]=y3_full[k];
                            d_found_result->threadId=(int)gid; d_found_result->iter=0;
                            __threadfence_system(); atomicExch(d_found_flag, FOUND_READY);
                        }
                    }
                    atomicAdd(hashes_accum, (unsigned long long)local_hashes);
                    return;
                }
            }

            // -Pi
            {
                uint64_t pxn[4], pyn[4];
#pragma unroll
                for (int j=0;j<4;++j) { pxn[j]=g_pGx[(size_t)i*4 + j]; pyn[j]=g_pGy[(size_t)i*4 + j]; }
                ModNeg256(pyn, pyn);

                uint64_t dy[4], lam[4], x3[4], s[4];
                ModSub256(dy, pyn, Y);
                _ModMult(lam, dy, dx);
                _ModSqr(x3, lam);
                ModSub256(x3, x3, X);
                ModSub256(x3, x3, pxn);
                ModSub256(s, X, x3);
                _ModMult(s, s, lam);
                uint8_t odd; ModSub256isOdd(s, Y, &odd);

                uint8_t h20[20]; getHash160_33_from_limbs(odd?0x03:0x02, x3, h20);
                ++local_hashes;
#if DEBUG_BATCH_CHECK
                ++tested;
#endif
                bool pref = (*(const uint32_t*)h20 == c_target_prefix);
                if (__any_sync(full_mask, pref)) {
                    if (pref && hash160_matches_prefix_then_full(h20, c_target_hash160, c_target_prefix)) {
                        if (atomicCAS(d_found_flag, FOUND_NONE, FOUND_LOCK) == FOUND_NONE) {
                            uint64_t fs[4];
#pragma unroll
                            for (int k=0;k<4;++k) fs[k]=S[k];
                            uint64_t sub=(uint64_t)(i+1);
#pragma unroll
                            for (int k=0;k<4 && sub;++k){ uint64_t old=fs[k]; fs[k]=old-sub; sub=(old<sub)?1ull:0ull; }
#pragma unroll
                            for (int k=0;k<4;++k){ d_found_result->scalar[k]=fs[k]; d_found_result->Rx[k]=x3[k]; }
                            uint64_t y3_full[4]; ModSub256(y3_full, s, Y);
#pragma unroll
                            for (int k=0;k<4;++k) d_found_result->Ry[k]=y3_full[k];
                            d_found_result->threadId=(int)gid; d_found_result->iter=0;
                            __threadfence_system(); atomicExch(d_found_flag, FOUND_READY);
                        }
                    }
                    atomicAdd(hashes_accum, (unsigned long long)local_hashes);
                    return;
                }
            }

            uint64_t tmpdx[4];
#pragma unroll
            for (int j=0;j<4;++j) tmpdx[j]=g_pGx[(size_t)i*4 + j];
            ModSub256(tmpdx, tmpdx, X);
            _ModMult(inverse, tmpdx);
        }

        // i=half-1: сначала -P(half-1)
        {
            int i = half-1;
            uint64_t dx[4]; _ModMult(dx, subp[i], inverse);

            uint64_t pxn[4], pyn[4];
#pragma unroll
            for (int j=0;j<4;++j) { pxn[j]=g_pGx[(size_t)i*4 + j]; pyn[j]=g_pGy[(size_t)i*4 + j]; }
            ModNeg256(pyn, pyn);

            uint64_t dy[4], lam[4], x3[4], s[4];
            ModSub256(dy, pyn, Y);
            _ModMult(lam, dy, dx);
            _ModSqr(x3, lam);
            ModSub256(x3, x3, X);
            ModSub256(x3, x3, pxn);
            ModSub256(s, X, x3);
            _ModMult(s, s, lam);
            uint8_t odd; ModSub256isOdd(s, Y, &odd);

            uint8_t h20[20]; getHash160_33_from_limbs(odd?0x03:0x02, x3, h20);
            ++local_hashes;
#if DEBUG_BATCH_CHECK
            ++tested;
#endif
            bool pref = (*(const uint32_t*)h20 == c_target_prefix);
            if (__any_sync(full_mask, pref)) {
                if (pref && hash160_matches_prefix_then_full(h20, c_target_hash160, c_target_prefix)) {
                    if (atomicCAS(d_found_flag, FOUND_NONE, FOUND_LOCK) == FOUND_NONE) {
                        uint64_t fs[4];
#pragma unroll
                        for (int k=0;k<4;++k) fs[k]=S[k];
                        uint64_t sub=(uint64_t)(half);
#pragma unroll
                        for (int k=0;k<4 && sub;++k){ uint64_t old=fs[k]; fs[k]=old-sub; sub=(old<sub)?1ull:0ull; }
#pragma unroll
                        for (int k=0;k<4;++k){ d_found_result->scalar[k]=fs[k]; d_found_result->Rx[k]=x3[k]; }
                        uint64_t y3_full[4]; ModSub256(y3_full, s, Y);
#pragma unroll
                        for (int k=0;k<4;++k) d_found_result->Ry[k]=y3_full[k];
                        d_found_result->threadId=(int)gid; d_found_result->iter=0;
                        __threadfence_system(); atomicExch(d_found_flag, FOUND_READY);
                    }
                }
                atomicAdd(hashes_accum, (unsigned long long)local_hashes);
                return;
            }
        }

        // затем +P(half-1) — ОБЯЗАТЕЛЬНО, чтобы получить ровно B проверок
        {
            int i = half-1;
            uint64_t dx[4]; _ModMult(dx, subp[i], inverse);

            uint64_t px_i[4], py_i[4];
#pragma unroll
            for (int j=0;j<4;++j) { px_i[j]=g_pGx[(size_t)i*4 + j]; py_i[j]=g_pGy[(size_t)i*4 + j]; }

            uint64_t dy[4], lam[4], x3[4], s[4];
            ModSub256(dy, py_i, Y);
            _ModMult(lam, dy, dx);
            _ModSqr(x3, lam);
            ModSub256(x3, x3, X);
            ModSub256(x3, x3, px_i);
            ModSub256(s, X, x3);
            _ModMult(s, s, lam);
            uint8_t odd; ModSub256isOdd(s, Y, &odd);

            uint8_t h20[20]; getHash160_33_from_limbs(odd?0x03:0x02, x3, h20);
            ++local_hashes;
#if DEBUG_BATCH_CHECK
            ++tested;
#endif
            bool pref = (*(const uint32_t*)h20 == c_target_prefix);
            if (__any_sync(full_mask, pref)) {
                if (pref && hash160_matches_prefix_then_full(h20, c_target_hash160, c_target_prefix)) {
                    if (atomicCAS(d_found_flag, FOUND_NONE, FOUND_LOCK) == FOUND_NONE) {
                        uint64_t fs[4];
#pragma unroll
                        for (int k=0;k<4;++k) fs[k]=S[k];
                        uint64_t addv=(uint64_t)(half);
#pragma unroll
                        for (int k=0;k<4 && addv;++k){ uint64_t old=fs[k]; fs[k]=old+addv; addv=(fs[k]<old)?1ull:0ull; }
#pragma unroll
                        for (int k=0;k<4;++k){ d_found_result->scalar[k]=fs[k]; d_found_result->Rx[k]=x3[k]; }
                        uint64_t y3_full[4]; ModSub256(y3_full, s, Y);
#pragma unroll
                        for (int k=0;k<4;++k) d_found_result->Ry[k]=y3_full[k];
                        d_found_result->threadId=(int)gid; d_found_result->iter=0;
                        __threadfence_system(); atomicExch(d_found_flag, FOUND_READY);
                    }
                }
                atomicAdd(hashes_accum, (unsigned long long)local_hashes);
                return;
            }
        }

#if DEBUG_BATCH_CHECK
        if (gid == 0) printf("DEBUG full-batch tested=%u expected=%d\n", tested, B);
        tested = 0;
#endif

        // перенос якоря на B*G
        {
            uint64_t Jx[4], Jy[4];
#pragma unroll
            for (int j=0;j<4;++j) { Jx[j]=g_pGx[(size_t)(B-1)*4 + j]; Jy[j]=g_pGy[(size_t)(B-1)*4 + j]; }
            uint64_t dxJ[4], dyJ[4], lamJ[4], xJ[4], sJ[4];
            ModSub256(dxJ, Jx, X);
            uint64_t invJ[5]; for (int j=0;j<4;++j) invJ[j]=dxJ[j]; invJ[4]=0ull; _ModInv(invJ);
            ModSub256(dyJ, Jy, Y);
            _ModMult(lamJ, dyJ, invJ);
            _ModSqr(xJ, lamJ);
            ModSub256(xJ, xJ, X);
            ModSub256(xJ, xJ, Jx);
            ModSub256(sJ, X, xJ);
            _ModMult(sJ, sJ, lamJ);
            ModSub256(sJ, sJ, Y);
#pragma unroll
            for (int j=0;j<4;++j){ X[j]=xJ[j]; Y[j]=sJ[j]; }
        }

        // S += B; rem -= B
        {
            uint64_t addv=(uint64_t)B;
#pragma unroll
            for (int i=0;i<4 && addv;++i) { uint64_t old=S[i]; S[i]=old+addv; addv=(S[i]<old)?1ull:0ull; }
        }
        sub256_u64_inplace(rem, (uint64_t)B);

    } else {
        // ===== ХВОСТ или «первый запуск после anchor»: ровно effective_B точек (+Pk) =====
        // active = min(effective_B, rem)
        uint64_t rem64 = rem[0];
        uint32_t active = (rem[3]|rem[2]|rem[1]) ? effective_B
                                                 : (uint32_t)((rem64 < (uint64_t)effective_B) ? rem64 : (uint64_t)effective_B);
        if (active == 0u) {
#pragma unroll
            for (int i=0;i<4;++i){ Rx[gid*4+i]=X[i]; Ry[gid*4+i]=Y[i]; counts256[gid*4+i]=rem[i]; start_scalars[gid*4+i]=S[i]; }
            atomicAdd(hashes_accum, (unsigned long long)local_hashes);
            return;
        }

        // dx[0..active-1]=(pGx[k]-X), dx[last_idx]= (B*G).x - X
        uint64_t last_idx = active;
        uint64_t dx[MAX_BATCH_SIZE + 1][4];
        uint64_t prod[MAX_BATCH_SIZE + 1][4];

        for (uint32_t k=0;k<active;++k) {
#pragma unroll
            for (int j=0;j<4;++j) dx[k][j] = g_pGx[(size_t)k*4 + j];
            ModSub256(dx[k], dx[k], X);
        }
        uint64_t Jx[4], Jy[4];
#pragma unroll
        for (int j=0;j<4;++j) { Jx[j]=g_pGx[(size_t)(B-1)*4 + j]; Jy[j]=g_pGy[(size_t)(B-1)*4 + j]; }
#pragma unroll
        for (int j=0;j<4;++j) dx[last_idx][j] = Jx[j];
        ModSub256(dx[last_idx], dx[last_idx], X);

        // prod
#pragma unroll
        for (int j=0;j<4;++j) prod[0][j] = dx[0][j];
        for (uint32_t i=1;i<=last_idx;++i) { _ModMult(prod[i], prod[i-1], dx[i]); }

        uint64_t inv_total[5]; for (int j=0;j<4;++j) inv_total[j]=prod[last_idx][j]; inv_total[4]=0ull; _ModInv(inv_total);

        uint64_t inv_dx_last[4];
        if (last_idx > 0) {
            uint64_t t[4]; for (int j=0;j<4;++j) t[j]=prod[last_idx-1][j];
            _ModMult(inv_dx_last, inv_total, t);
        } else {
            for (int j=0;j<4;++j) inv_dx_last[j]=inv_total[j];
        }

        uint64_t acc_suffix[4] = { 1ull, 0ull, 0ull, 0ull };
        _ModMult(acc_suffix, acc_suffix, dx[last_idx]);

        // k=active-1..0: только +Pk
        for (int k=(int)active-1; k>=0; --k) {
            uint64_t left_prod[4];
            if (k>0) { for (int j=0;j<4;++j) left_prod[j]=prod[k-1][j]; }
            else { left_prod[0]=1ull; left_prod[1]=0ull; left_prod[2]=0ull; left_prod[3]=0ull; }

            uint64_t inv_dx_k[4], tmp[4];
            _ModMult(tmp, inv_total, left_prod);
            _ModMult(inv_dx_k, tmp, acc_suffix);

            // +Pk
            {
                uint64_t px_i[4], py_i[4];
#pragma unroll
                for (int j=0;j<4;++j) { px_i[j]=g_pGx[(size_t)k*4 + j]; py_i[j]=g_pGy[(size_t)k*4 + j]; }
                uint64_t dy[4], lam[4], x3[4], s[4];
                ModSub256(dy, py_i, Y);
                _ModMult(lam, dy, inv_dx_k);
                _ModSqr(x3, lam);
                ModSub256(x3, x3, X);
                ModSub256(x3, x3, px_i);
                ModSub256(s, X, x3);
                _ModMult(s, s, lam);
                uint8_t odd; ModSub256isOdd(s, Y, &odd);

                uint8_t h20[20]; getHash160_33_from_limbs(odd?0x03:0x02, x3, h20);
                ++local_hashes;
#if DEBUG_BATCH_CHECK
                ++tested;
#endif
                bool pref = (*(const uint32_t*)h20 == c_target_prefix);
                if (__any_sync(full_mask, pref)) {
                    if (pref && hash160_matches_prefix_then_full(h20, c_target_hash160, c_target_prefix)) {
                        if (atomicCAS(d_found_flag, FOUND_NONE, FOUND_LOCK) == FOUND_NONE) {
                            uint64_t fs[4];
#pragma unroll
                            for (int t=0;t<4;++t) fs[t]=S[t];
                            uint64_t addv=(uint64_t)(k+1);
#pragma unroll
                            for (int t=0;t<4 && addv;++t){ uint64_t old=fs[t]; fs[t]=old+addv; addv=(fs[t]<old)?1ull:0ull; }
#pragma unroll
                            for (int t=0;t<4;++t){ d_found_result->scalar[t]=fs[t]; d_found_result->Rx[t]=x3[t]; }
                            uint64_t y3_full[4]; ModSub256(y3_full, s, Y);
#pragma unroll
                            for (int t=0;t<4;++t) d_found_result->Ry[t]=y3_full[t];
                            d_found_result->threadId=(int)gid; d_found_result->iter=0;
                            __threadfence_system(); atomicExch(d_found_flag, FOUND_READY);
                        }
                    }
                    atomicAdd(hashes_accum, (unsigned long long)local_hashes);
                    return;
                }
            }

            _ModMult(acc_suffix, acc_suffix, dx[k]);
        }

#if DEBUG_BATCH_CHECK
        if (gid == 0) printf("DEBUG tail tested=%u expected=%u\n", tested, active);
        tested = 0;
#endif

        // перенос якоря на active*G
        {
            uint64_t GxA[4], GyA[4];
#pragma unroll
            for (int j=0;j<4;++j) { GxA[j]=g_pGx[(size_t)(active-1)*4 + j]; GyA[j]=g_pGy[(size_t)(active-1)*4 + j]; }
            uint64_t dxA[4], dyA[4], lamA[4], xA[4], sA[4];
            ModSub256(dxA, GxA, X);
            uint64_t invA[5]; for (int j=0;j<4;++j) invA[j]=dxA[j]; invA[4]=0ull; _ModInv(invA);
            ModSub256(dyA, GyA, Y);
            _ModMult(lamA, dyA, invA);
            _ModSqr(xA, lamA);
            ModSub256(xA, xA, X);
            ModSub256(xA, xA, GxA);
            ModSub256(sA, X, xA);
            _ModMult(sA, sA, lamA);
            ModSub256(sA, sA, Y);
#pragma unroll
            for (int j=0;j<4;++j){ X[j]=xA[j]; Y[j]=sA[j]; }
        }

        // S += active; rem -= active
        {
            uint64_t addv=(uint64_t)active;
#pragma unroll
            for (int i=0;i<4 && addv;++i) { uint64_t old=S[i]; S[i]=old+addv; addv=(S[i]<old)?1ull:0ull; }
        }
        sub256_u64_inplace(rem, (uint64_t)active);
    }

    // write back
#pragma unroll
    for (int i=0;i<4;++i) {
        Rx[gid*4+i] = X[i];
        Ry[gid*4+i] = Y[i];
        counts256[gid*4+i] = rem[i];
        start_scalars[gid*4+i] = S[i];
    }
    atomicAdd(hashes_accum, (unsigned long long)local_hashes);
}

extern bool hexToLE64(const std::string& h_in, uint64_t w[4]);
extern bool hexToHash160(const std::string& h, uint8_t hash160[20]);
extern std::string formatHex256(const uint64_t limbs[4]);
extern long double ld_from_u256(const uint64_t v[4]);

int main(int argc, char** argv) {
    std::string target_hash_hex, range_hex, address_b58;
    uint32_t runtime_points_batch_size = 512;
    uint32_t runtime_batches_per_sm    = 256;
    bool grid_provided=false;

    auto parse_grid = [](const std::string& s, uint32_t& a_out, uint32_t& b_out)->bool {
        size_t comma = s.find(',');
        if (comma == std::string::npos) return false;
        auto trim = [](std::string& z){
            size_t p1 = z.find_first_not_of(" \t");
            size_t p2 = z.find_last_not_of(" \t");
            if (p1 == std::string::npos) { z.clear(); return; }
            z = z.substr(p1, p2 - p1 + 1);
        };
        std::string a_str = s.substr(0, comma);
        std::string b_str = s.substr(comma + 1);
        trim(a_str); trim(b_str);
        if (a_str.empty() || b_str.empty()) return false;
        char* endp=nullptr;
        unsigned long aa = std::strtoul(a_str.c_str(), &endp, 10); if (*endp) return false;
        endp=nullptr;
        unsigned long bb = std::strtoul(b_str.c_str(), &endp, 10); if (*endp) return false;
        if (aa == 0ul || bb == 0ul) return false;
        if (aa > (1ul<<20) || bb > (1ul<<20)) return false;
        a_out=(uint32_t)aa; b_out=(uint32_t)bb; return true;
    };

    for (int i=1;i<argc;++i) {
        std::string arg = argv[i];
        if      (arg == "--target-hash160" && i + 1 < argc) target_hash_hex = argv[++i];
        else if (arg == "--address"        && i + 1 < argc) address_b58     = argv[++i];
        else if (arg == "--range"          && i + 1 < argc) range_hex       = argv[++i];
        else if (arg == "--grid"           && i + 1 < argc) {
            uint32_t a=0,b=0;
            if (!parse_grid(argv[++i], a, b)) { std::cerr<<"Error: --grid expects \"A,B\".\n"; return EXIT_FAILURE; }
            runtime_points_batch_size = a;
            runtime_batches_per_sm    = b;
            grid_provided = true;
        }
    }

    if (range_hex.empty() || (target_hash_hex.empty() && address_b58.empty())) {
        std::cerr << "Usage: " << argv[0]
                  << " --range <start_hex>:<end_hex> (--address <base58> | --target-hash160 <hash160_hex>) [--grid A,B]\n";
        return EXIT_FAILURE;
    }
    if (!target_hash_hex.empty() && !address_b58.empty()) {
        std::cerr << "Error: provide either --address or --target-hash160, not both.\n"; return EXIT_FAILURE;
    }

    size_t colon_pos = range_hex.find(':');
    if (colon_pos == std::string::npos) { std::cerr << "Error: range format must be start:end\n"; return EXIT_FAILURE; }
    std::string start_hex = range_hex.substr(0, colon_pos);
    std::string end_hex   = range_hex.substr(colon_pos + 1);

    uint64_t range_start[4]{0}, range_end[4]{0};
    if (!hexToLE64(start_hex, range_start) || !hexToLE64(end_hex, range_end)) {
        std::cerr << "Error: invalid range hex\n"; return EXIT_FAILURE;
    }

    uint8_t target_hash160[20];
    if (!address_b58.empty()) {
        if (!decode_p2pkh_address(address_b58, target_hash160)) {
            std::cerr << "Error: invalid P2PKH address\n"; return EXIT_FAILURE;
        }
    } else {
        if (!hexToHash160(target_hash_hex, target_hash160)) {
            std::cerr << "Error: invalid target hash160 hex\n"; return EXIT_FAILURE;
        }
    }

    // range_len = end - start + 1
    uint64_t range_len[4]; sub256(range_end, range_start, range_len); add256_u64(range_len, 1ull, range_len);

    // проверки степени двойки и выравнивания старта
    auto is_zero_256 = [](const uint64_t a[4])->bool { return (a[0]|a[1]|a[2]|a[3])==0ull; };
    auto is_power_of_two_256 = [&](const uint64_t a[4])->bool {
        if (is_zero_256(a)) return false;
        uint64_t am1[4]; uint64_t borrow=1ull;
        for (int i=0;i<4;++i){ uint64_t v=a[i]-borrow; borrow=(a[i]<borrow)?1ull:0ull; am1[i]=v; if (!borrow && i+1<4){ for(int k=i+1;k<4;++k) am1[k]=a[k]; break; } }
        uint64_t and0=a[0]&am1[0],and1=a[1]&am1[1],and2=a[2]&am1[2],and3=a[3]&am1[3];
        return (and0|and1|and2|and3)==0ull;
    };
    if (!is_power_of_two_256(range_len)) { std::cerr<<"Error: range length must be a power of two.\n"; return EXIT_FAILURE; }
    uint64_t len_minus1[4]; { uint64_t borrow=1ull; for (int i=0;i<4;++i){ uint64_t v=range_len[i]-borrow; borrow=(range_len[i]<borrow)?1ull:0ull; len_minus1[i]=v; if(!borrow && i+1<4){ for(int k=i+1;k<4;++k) len_minus1[k]=range_len[k]; break; } } }
    { uint64_t and0=range_start[0]&len_minus1[0],and1=range_start[1]&len_minus1[1],and2=range_start[2]&len_minus1[2],and3=range_start[3]&len_minus1[3];
      if ((and0|and1|and2|and3)!=0ull) { std::cerr<<"Error: start must be aligned to the range length.\n"; return EXIT_FAILURE; } }

    // CUDA init
    int device=0; hipDeviceProp_t prop{}; if (hipGetDevice(&device)!=hipSuccess || hipGetDeviceProperties(&prop, device)!=hipSuccess) {
        std::cerr<<"CUDA init error\n"; return EXIT_FAILURE;
    }
    int threadsPerBlock=256;
    if (threadsPerBlock > (int)prop.maxThreadsPerBlock) threadsPerBlock=prop.maxThreadsPerBlock;
    if (threadsPerBlock < 32) threadsPerBlock=32;

    // подбор threadsTotal
    uint64_t bytesPerThread = 2ull*4ull*sizeof(uint64_t);
    size_t totalGlobalMem = prop.totalGlobalMem;
    uint64_t reserveBytes = 64ull*1024ull*1024ull;
    uint64_t usableMem = (totalGlobalMem>reserveBytes) ? (totalGlobalMem-reserveBytes) : (totalGlobalMem/2);
    uint64_t maxThreadsByMem = usableMem / bytesPerThread;

    uint64_t q_div_batch[4], r_div_batch=0ull;
    divmod_256_by_u64_safe(range_len, (uint64_t)runtime_points_batch_size, q_div_batch, r_div_batch);
    if (r_div_batch != 0ull) { std::cerr<<"Error: range length must be divisible by batch size.\n"; return EXIT_FAILURE; }

    bool q_fits_u64 = (q_div_batch[3]|q_div_batch[2]|q_div_batch[1])==0ull;
    uint64_t q_u64  = q_fits_u64 ? q_div_batch[0] : UINT64_MAX;
    auto pick_threads_total = [&](uint64_t upper)->uint64_t {
        if (upper < (uint64_t)threadsPerBlock) return 0ull;
        uint64_t t = upper - (upper % (uint64_t)threadsPerBlock);
        if (!q_fits_u64) return t;
        uint64_t q = q_u64;
        while (t >= (uint64_t)threadsPerBlock) {
            if ((q % t) == 0ull) return t;
            t -= (uint64_t)threadsPerBlock;
        }
        return 0ull;
    };
    uint64_t upper = maxThreadsByMem;
    uint64_t userUpper = (uint64_t)prop.multiProcessorCount * (uint64_t)runtime_batches_per_sm * (uint64_t)threadsPerBlock;
    if (userUpper == 0ull) userUpper = UINT64_MAX;
    if (q_fits_u64 && q_u64 < upper) upper = q_u64;
    if (userUpper   < upper)         upper = userUpper;

    uint64_t threadsTotal = pick_threads_total(upper);
    if (threadsTotal == 0ull) { std::cerr<<"Error: failed to pick threadsTotal.\n"; return EXIT_FAILURE; }
    int blocks = (int)(threadsTotal / (uint64_t)threadsPerBlock);

    // per-thread count
    uint64_t per_thread_count[4], r_u64=0ull;
    divmod_256_by_u64_safe(range_len, threadsTotal, per_thread_count, r_u64);
    if (r_u64 != 0ull) { std::cerr<<"Internal error: per_thread_count not integer.\n"; return EXIT_FAILURE; }

    // Хост-буферы
    uint64_t* h_counts256     = new uint64_t[threadsTotal * 4];
    uint64_t* h_start_scalars = new uint64_t[threadsTotal * 4];
    {
        for (uint64_t i=0;i<threadsTotal;++i) {
            h_counts256[i*4+0]=per_thread_count[0];
            h_counts256[i*4+1]=per_thread_count[1];
            h_counts256[i*4+2]=per_thread_count[2];
            h_counts256[i*4+3]=per_thread_count[3];
        }
        uint64_t cur[4] = { range_start[0], range_start[1], range_start[2], range_start[3] };
        for (uint64_t i=0;i<threadsTotal;++i) {
            h_start_scalars[i*4+0]=cur[0];
            h_start_scalars[i*4+1]=cur[1];
            h_start_scalars[i*4+2]=cur[2];
            h_start_scalars[i*4+3]=cur[3];
            uint64_t next[4]; add256(cur, per_thread_count, next);
            cur[0]=next[0]; cur[1]=next[1]; cur[2]=next[2]; cur[3]=next[3];
        }
    }

    // target prefix/hash
    {
        uint32_t prefix_le = (uint32_t)target_hash160[0]
                           | ((uint32_t)target_hash160[1] << 8)
                           | ((uint32_t)target_hash160[2] << 16)
                           | ((uint32_t)target_hash160[3] << 24);
        hipMemcpyToSymbol(HIP_SYMBOL(c_target_prefix), &prefix_le, sizeof(prefix_le));
        hipMemcpyToSymbol(HIP_SYMBOL(c_target_hash160), target_hash160, 20);
    }

    // Device buffers
    uint64_t *d_start_scalars=nullptr, *d_Px=nullptr, *d_Py=nullptr, *d_Rx=nullptr, *d_Ry=nullptr, *d_counts256=nullptr;
    int *d_found_flag=nullptr; FoundResult *d_found_result=nullptr; unsigned long long *d_hashes_accum=nullptr;
    hipMalloc(&d_start_scalars, threadsTotal * 4 * sizeof(uint64_t));
    hipMalloc(&d_Px, threadsTotal * 4 * sizeof(uint64_t));
    hipMalloc(&d_Py, threadsTotal * 4 * sizeof(uint64_t));
    hipMalloc(&d_Rx, threadsTotal * 4 * sizeof(uint64_t));
    hipMalloc(&d_Ry, threadsTotal * 4 * sizeof(uint64_t));
    hipMalloc(&d_counts256, threadsTotal * 4 * sizeof(uint64_t));
    hipMalloc(&d_found_flag, sizeof(int));
    hipMalloc(&d_found_result, sizeof(FoundResult));
    hipMalloc(&d_hashes_accum, sizeof(unsigned long long));
    hipMemcpy(d_start_scalars, h_start_scalars, threadsTotal * 4 * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_counts256,     h_counts256,     threadsTotal * 4 * sizeof(uint64_t), hipMemcpyHostToDevice);
    {
        int zero = FOUND_NONE; unsigned long long zero64=0ull;
        hipMemcpy(d_found_flag, &zero, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_hashes_accum, &zero64, sizeof(unsigned long long), hipMemcpyHostToDevice);
    }

    // Начальные XY = S*G
    {
        int blocks_scal = (int)((threadsTotal + threadsPerBlock - 1) / threadsPerBlock);
        scalarMulKernelBase<<<blocks_scal, threadsPerBlock>>>(d_start_scalars, d_Px, d_Py, (int)threadsTotal);
        hipDeviceSynchronize();
    }

    // Предрасчёт 1..B·G → g_pGx/g_pGy
    {
        const uint32_t B = runtime_points_batch_size;
        uint64_t *d_pGx=nullptr, *d_pGy=nullptr; hipMalloc(&d_pGx, (size_t)B * 4 * sizeof(uint64_t)); hipMalloc(&d_pGy, (size_t)B * 4 * sizeof(uint64_t));
        uint64_t* h_scal = (uint64_t*)malloc((size_t)B * 4 * sizeof(uint64_t));
        std::memset(h_scal, 0, (size_t)B * 4 * sizeof(uint64_t));
        for (uint32_t k=0;k<B;++k) h_scal[(size_t)k*4 + 0] = (uint64_t)(k + 1);
        uint64_t *d_pG_scalars=nullptr; hipMalloc(&d_pG_scalars, (size_t)B * 4 * sizeof(uint64_t));
        hipMemcpy(d_pG_scalars, h_scal, (size_t)B * 4 * sizeof(uint64_t), hipMemcpyHostToDevice);
        int blocks_scal = (int)((B + threadsPerBlock - 1) / threadsPerBlock);
        scalarMulKernelBase<<<blocks_scal, threadsPerBlock>>>(d_pG_scalars, d_pGx, d_pGy, (int)B);
        hipDeviceSynchronize();
        hipMemcpyToSymbol(HIP_SYMBOL(g_pGx), d_pGx, (size_t)B * 4 * sizeof(uint64_t), 0, hipMemcpyDeviceToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(g_pGy), d_pGy, (size_t)B * 4 * sizeof(uint64_t), 0, hipMemcpyDeviceToDevice);
        hipFree(d_pG_scalars); free(h_scal); hipFree(d_pGx); hipFree(d_pGy);
    }

    // Info
    size_t freeB=0,totalB=0; hipMemGetInfo(&freeB,&totalB);
    double util = totalB ? (double)(totalB-freeB)*100.0/(double)totalB : 0.0;
    std::cout << "======== PrePhase: GPU Information ====================\n";
    std::cout << std::left << std::setw(20) << "Device"            << " : " << prop.name << " (compute " << prop.major << "." << prop.minor << ")\n";
    std::cout << std::left << std::setw(20) << "SM"                << " : " << prop.multiProcessorCount << "\n";
    std::cout << std::left << std::setw(20) << "ThreadsPerBlock"   << " : " << threadsPerBlock << "\n";
    std::cout << std::left << std::setw(20) << "Blocks"            << " : " << blocks << "\n";
    std::cout << std::left << std::setw(20) << "Points batch size" << " : " << runtime_points_batch_size << "\n";
    std::cout << std::left << std::setw(20) << "Memory utilization"<< " : " << std::fixed << std::setprecision(1) << util << "% ("
              << human_bytes((double)(totalB-freeB)) << " / " << human_bytes((double)totalB) << ")\n";
    std::cout << "------------------------------------------------------- \n";
    std::cout << std::left << std::setw(20) << "Total threads"     << " : " << threadsTotal << "\n\n";
    std::cout << "======== Phase-1: BruteForce (1 batch / launch, ±, 1 inv) =====\n";

    auto t0 = std::chrono::high_resolution_clock::now();
    auto tLast = t0; unsigned long long lastHashes=0ull;
    bool stop_all=false; bool first_launch = true;

    while (!stop_all) {
        kernel_one_batch_pm_both<<<blocks, threadsPerBlock>>>(
            d_Px, d_Py, d_Rx, d_Ry,
            d_start_scalars, d_counts256,
            threadsTotal,
            runtime_points_batch_size,
            first_launch ? 1 : 0,
            d_found_flag, d_found_result,
            d_hashes_accum
        );
        hipError_t ke = hipGetLastError();
        if (ke != hipSuccess) { std::cerr << "Kernel error: " << hipGetErrorString(ke) << "\n"; return EXIT_FAILURE; }
        hipDeviceSynchronize();

        auto now = std::chrono::high_resolution_clock::now();
        double dt = std::chrono::duration<double>(now - tLast).count();
        if (dt >= 1.0) {
            unsigned long long h_hashes = 0ull; hipMemcpy(&h_hashes, d_hashes_accum, sizeof(unsigned long long), hipMemcpyDeviceToHost);
            double delta = (double)(h_hashes - lastHashes);
            double mkeys = delta / (dt * 1e6);
            double elapsed = std::chrono::duration<double>(now - t0).count();
            long double total_keys_ld = ld_from_u256(range_len);
            long double prog = total_keys_ld > 0.0L ? ((long double)h_hashes / total_keys_ld) * 100.0L : 0.0L;
            if (prog > 100.0L) prog = 100.0L;
            std::cout << "\rTime: " << std::fixed << std::setprecision(1) << elapsed
                      << " s | Speed: " << std::fixed << std::setprecision(1) << mkeys
                      << " Mkeys/s | Count: " << h_hashes
                      << " | Progress: " << std::fixed << std::setprecision(2) << (double)prog << " %";
            std::cout.flush();
            lastHashes = h_hashes; tLast = now;
        }

        int h_found=0; hipMemcpy(&h_found, d_found_flag, sizeof(int), hipMemcpyDeviceToHost);
        if (h_found == FOUND_READY) stop_all = true;

        std::swap(d_Px, d_Rx); std::swap(d_Py, d_Ry);
        first_launch = false;

        if (stop_all) break;
        // (Останов по исчерпанию диапазона можно реализовать анализом counts256; опущено для краткости.)
    }
    std::cout << "\n";

    int h_found_flag=0; hipMemcpy(&h_found_flag, d_found_flag, sizeof(int), hipMemcpyDeviceToHost);
    if (h_found_flag == FOUND_READY) {
        FoundResult host_result{}; hipMemcpy(&host_result, d_found_result, sizeof(FoundResult), hipMemcpyDeviceToHost);
        std::cout << "\n======== FOUND MATCH! =================================\n";
        std::cout << "Private Key   : " << formatHex256(host_result.scalar) << "\n";
        std::cout << "Public Key    : " << formatCompressedPubHex(host_result.Rx, host_result.Ry) << "\n";
    }

    hipFree(d_start_scalars); hipFree(d_Px); hipFree(d_Py); hipFree(d_Rx); hipFree(d_Ry);
    hipFree(d_counts256); hipFree(d_found_flag); hipFree(d_found_result); hipFree(d_hashes_accum);
    delete[] h_start_scalars; delete[] h_counts256;

    return 0;
}

